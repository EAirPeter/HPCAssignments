#include "hip/hip_runtime.h"
#define NDEBUG

#include <algorithm>
#include <cmath>
#include <random>

#include <omp.h>

#include "utils.h"

using Num = double;

constexpr size_t alignment = 128;

Num* allocNum(size_t n) {
  if (auto res = aligned_alloc(alignment, n * sizeof(Num)))
    return (Num*) __builtin_assume_aligned(res, alignment);
  std::fprintf(stderr, "Allocation of %zu bytes failed", n * sizeof(Num));
  std::exit(EXIT_FAILURE);
}


void dotProdRef(Num* __restrict__ pz,
  const Num* __restrict__ x, const Num* __restrict__ y, long n)
{
  x = (const Num*) __builtin_assume_aligned(x, alignment);
  y = (const Num*) __builtin_assume_aligned(y, alignment);

  Num z = 0;
  for (auto i = 0l; i < n; ++i)
    z += x[i] * y[i];

  *pz = z;
}

void dotProdOmp(Num* __restrict__ pz,
  const Num* __restrict__ x, const Num* __restrict__ y, long n)
{
  x = (const Num*) __builtin_assume_aligned(x, alignment);
  y = (const Num*) __builtin_assume_aligned(y, alignment);

  Num z = 0;
# pragma omp parallel for reduction(+: z) schedule(static)
  for (auto i = 0l; i < n; ++i)
    z += x[i] * y[i];

  *pz = z;
}

template<long blkSize>
__global__
void dotProdKernel(Num* __restrict__ z,
  const Num* __restrict__ x, const Num* __restrict__ y, long n)
{
  extern __shared__ Num tmp[];

  auto tid = threadIdx.x;
  auto i = tid + blockIdx.x * blkSize * 2;
  auto stride = gridDim.x * blkSize * 2;

  Num res = 0;

  while (i < n) {
    res += x[i] * y[i];
    if (i + blkSize < n)
      res += x[i + blkSize] * y[i + blkSize];
    i += stride;
  }

  tmp[tid] = res;
  __syncthreads();

  auto h = blkSize / 2;

  while (h > 32) {
    if (tid < h)
      tmp[tid] = res += tmp[tid + h];
    __syncthreads();
    h >>= 1;
  }

  if (tid < 32) {
    while (h) {
      tmp[tid] = res += tmp[tid + h];
      __syncthreads();
      h >>= 1;
    }
  }

  if (!tid)
    z[blockIdx.x] = res;
}

template<long grdSize, long blkSize>
void dotProdGpu(Num* z, Num* __restrict__ d_z,
  const Num* __restrict__ d_x, const Num* __restrict__ d_y, long n)
{
  Num tmp[grdSize] alignas(alignment);
  auto tmpSize = (blkSize < 64 ? 64 : blkSize) * sizeof(Num);
  auto nBlk = std::min(grdSize, (n + blkSize * 2 - 1) / (blkSize * 2));
  dotProdKernel<blkSize><<<nBlk, blkSize, tmpSize>>>(d_z, d_x, d_y, n);
  checkCuda(hipMemcpy(tmp, d_z, nBlk * sizeof(Num), hipMemcpyDeviceToHost));
  Num res = 0;
  for (auto i = 0l; i < nBlk; ++i)
    res += tmp[i];
  *z = res;
}

void processData(const char* name, double time,
  Num z, Num zRef, long n, long nIter)
{
  auto error = fabs(z - zRef);
  auto bandwidth = nIter * n * 2 * sizeof(Num) * 1e-9 / time;
  printf("%12s %12.6f %18.6f %16e\n", name, time, bandwidth, error);
}

int main() {
  constexpr long grdSize = 128;
  constexpr long blkSize = 128;
  constexpr long nIter = 20;
  constexpr long n = 10000000;

  hipDeviceProp_t prop;
  checkCuda(hipGetDeviceProperties(&prop, 0));
  checkCuda(hipSetDevice(0));
  printf("Device: %s\n", prop.name);
  printf("Vector Dimension: %d\n", n);
  
  // Data generation
  auto x = allocNum(n);
  auto y = allocNum(n);

  std::mt19937_64 rand{std::random_device{}()};
  auto randpm1 = [&] {
    return std::uniform_real_distribution<Num>(-1, 1)(rand);
  };

  std::generate(x, x + n, randpm1);
  std::generate(y, y + n, randpm1);

  // GPU memory initialization
  size_t memSize = n * sizeof(Num);
  Num *d_x, *d_y, *d_z;
  checkCuda(hipMalloc(&d_x, memSize));
  checkCuda(hipMalloc(&d_y, memSize));
  checkCuda(hipMalloc(&d_z, memSize));

  checkCuda(hipMemcpy(d_x, x, memSize, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_y, y, memSize, hipMemcpyHostToDevice));

  Timer t;
  
  printf("\n%12s %12s %18s %16s\n",
    "Routine", "Time (s)", "Bandwidth (GB/s)", "Error");

  // Results
  Num zRef, zOmp, zGpu;

  // Serial/reference
  dotProdRef(&zRef, x, y, n);
  t.tic();
  for (volatile auto iter = 0l; iter < nIter; ++iter)
    dotProdRef(&zRef, x, y, n);
  processData("dotProdRef", t.toc(), zRef, zRef, n, nIter);

  // OpenMP
  dotProdOmp(&zOmp, x, y, n);
  t.tic();
  for (volatile auto iter = 0l; iter < nIter; ++iter)
    dotProdOmp(&zOmp, x, y, n);
  processData("dotProdOmp", t.toc(), zOmp, zRef, n, nIter);

  // GPU/CUDA
  dotProdGpu<grdSize, blkSize>(&zGpu, d_z, d_x, d_y, n);
  t.tic();
  for (volatile auto iter = 0l; iter < nIter; ++iter)
    dotProdGpu<grdSize, blkSize>(&zGpu, d_z, d_x, d_y, n);
  auto time = t.toc();
  processData("dotProdGpu", time, zGpu, zRef, n, nIter);

  checkCuda(hipFree(d_x));
  checkCuda(hipFree(d_y));
  checkCuda(hipFree(d_z));

  free(x);
  free(y);
  return 0;
}
